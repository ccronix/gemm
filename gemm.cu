#include <omp.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

#define SIZE 1000
#define SUB_MAT 16


/*
    1, 2, 3     1, 2      22, 28
    4, 5, 6     3, 4      49, 64 
                5, 6
*/


__host__
void cpu_gemm(float* matrix_a, float* matrix_b, float* result, int a_rows, int a_cols, int b_cols)
{
    for (int i = 0; i < a_rows; i++) {
        // printf("\rmul percentage: %.3f%%", 100. * (i + 1) / a_rows);
        for (int j = 0; j < b_cols; j++) {
            for (int k = 0; k < a_cols; k++) {
                result[i * a_rows + j] += matrix_a[i * a_cols + k] * matrix_b[k * b_cols + j];
            }
        }
    }
    // printf("\n");
}


__global__
void cuda_gemm_kernel(float* matrix_a, float* matrix_b, float* result, int a_rows, int a_cols, int b_cols)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= a_rows || j >= b_cols) return;
    
    float sum = 0;
    for (int k = 0; k < a_cols; k++) {
        sum += matrix_a[i * a_cols + k] * matrix_b[k * b_cols + j];
    }
    result[i * a_rows + j] = sum;
}


void printm(float* matrix, int matrix_rows, int matrix_cols)
{
    float curr_val = 0;
    for (int i = 0; i < matrix_rows; i++) {
        for (int j = 0; j < matrix_cols; j++) {
            curr_val = matrix[i * matrix_cols + j];
            printf("%.3f ", curr_val);
        }
        printf("\n");
    }
    printf("\n");
}


void random_matrix(float* genm)
{
    for (int i = 0; i < SIZE * SIZE; i++) {
        float random_float = (float) rand() / RAND_MAX;
        genm[i] = random_float;
    }
}

void gen_test_matrix(float** test_mat, int array_len)
{
    for (int i = 0; i < array_len; i++){
        float* curr_mat = (float*) malloc(SIZE * SIZE * sizeof(float));
        random_matrix(curr_mat);
        test_mat[i] = curr_mat;
    }
}


__host__
void cuda_benchmark(int suite_length)
{
    printf("alloating host memory...\n");

    float** groupa = (float**) malloc(suite_length * sizeof(float*));
    float** groupb = (float**) malloc(suite_length * sizeof(float*));
    float** groupc = (float**) malloc(suite_length * sizeof(float*));

    printf("host memory done, generating test suite...\n");

    gen_test_matrix(groupa, suite_length);
    gen_test_matrix(groupb, suite_length);

    printf("test suite done, allocating result memory...\n");

    for (int i = 0; i < suite_length; i++) {
        float* curr_mem = (float*) malloc(SIZE * SIZE * sizeof(float));
        groupc[i] = curr_mem;
    }

    printf("result memory done, allocating gpu memory...\n");

    float* gpu_groupa, * gpu_groupb, * gpu_groupc;
    hipMalloc(& gpu_groupa, SIZE * SIZE * sizeof(float));
    hipMalloc(& gpu_groupb, SIZE * SIZE * sizeof(float));
    hipMalloc(& gpu_groupc, SIZE * SIZE * sizeof(float));

    printf("allocate gpu memory done, starting kernel...\n");

    double start = clock();

    for (int i = 0; i < suite_length; i++) {
        hipMemcpy(gpu_groupa, groupa[i], SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(gpu_groupb, groupb[i], SIZE * SIZE * sizeof(float), hipMemcpyHostToDevice);

        const int dim = 8;

        dim3 dimGrid(SIZE / dim, SIZE / dim, 1); 
        dim3 dimBlock(dim, dim, 1);

        cuda_gemm_kernel<<<dimGrid, dimBlock>>>(gpu_groupa, gpu_groupb, gpu_groupc, SIZE, SIZE, SIZE);
        hipDeviceSynchronize();

        hipMemcpy(groupc[i], gpu_groupc, SIZE * SIZE * sizeof(float), hipMemcpyDeviceToHost);
    }

    double end = clock();
    double running_secs = (end - start) / CLOCKS_PER_SEC;

    printf("%d %dx%d gemm end, using %f secs.\n", suite_length, SIZE, SIZE, running_secs);

    printf("performance: %.3f GFLOPS.\n", suite_length * 2 / running_secs);
    printf("kernel done.\n");

    hipFree(gpu_groupa);
    hipFree(gpu_groupb);
    hipFree(gpu_groupc);

}


int main(int argc, char* argv[])
{
    int test_len = atoi(argv[1]);
    cuda_benchmark(test_len);
    return 0;
}